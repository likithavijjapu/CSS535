#include "hip/hip_runtime.h"
# include <time.h>

__global__ void add( int *a , int *b , int *c)
{
	clock_t start_time = clock(); 
	c[blockIdx.x] = a[blockIdx.x] +b [blockIdx.x];
	/*hipDeviceSynchronize(); */
	clock_t stop_time = clock();
	printf("time=%d\n", (stop_time - start_time) );
}

# define N 512

int main(void){
	int *a , *b , *c;
	int *d_a , *d_b , *d_c;

	int size = N * sizeof(int);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
