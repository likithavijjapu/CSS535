#include "hip/hip_runtime.h"
# include <time.h>
# include <math.h>
# include <stdio.h>

__global__ void add( int *a , int *b , int *c)
{
	
	c[blockIdx.x] = a[blockIdx.x] +b[blockIdx.x];
	
}

//# define N 125

void random_ints(int* a, int h)
{

}

int main(void){
	int N;
	printf("\"Hello Vector !\"\n enter size of vector\n");
	scanf("%d",&N);
	int a[N],b[N],c[N];
	int *d_a,*d_b,*d_c;

	int size = N * sizeof(int);
	
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	for ( int i=0;i<=N;i++)
  		{
    		a[i]=i+2;
    		b[i]=i+3;
   		c[i]=0;
  		}

	
	
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
	clock_t start_time = clock(); 
	add<<<N,1>>>(d_a, d_b, d_c);
	hipDeviceSynchronize(); 
	clock_t stop_time = clock();
	int time =stop_time - start_time;
	printf("time=%d\n", time);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("c=");
	for(int i=0;i<N;i++){
	
	printf("%d+",c[i]);}
	printf("\n");

	
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
