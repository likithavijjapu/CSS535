
#include <hip/hip_runtime.h>
# include <time.h>
# include <math.h>
# include <stdio.h>

__global__ void add( int *a , int *b , int *c)
{
	clock_t start_time = clock(); 
	c[blockIdx.x] = a[blockIdx.x] +b[blockIdx.x];
	/*cudaThreadSynchronize(); */
	clock_t stop_time = clock();
	printf("time=%d\n", (stop_time - start_time) );
}

# define N 5

void random_ints(int* a, int h)
{

}

int main(void){
	int a[N],b[N],c[N];
	int *d_a,*d_b,*d_c;

	int size = N * sizeof(int);
	
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	for ( int i=0;i<=N;i++)
  		{
    		a[i]=i+2;
    		b[i]=i+3;
   		c[i]=0;
  		}

	
	
	printf("a=%d\n",a);
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("sum=%d\n",c);

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
