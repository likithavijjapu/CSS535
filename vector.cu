
#include <hip/hip_runtime.h>
# include <time.h>
# include <math.h>
# include <stdio.h>

__global__ void add( int *a , int *b , int *c)
{
	clock_t start_time = clock(); 
	c[blockIdx.x] = a[blockIdx.x] +b[blockIdx.x];
	/*cudaThreadSynchronize(); */
	clock_t stop_time = clock();
	printf("time=%d\n", (stop_time - start_time) );
}

# define N 5

void random_ints(int* a, int h)
{

}

int main(void){
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;

	int size = N * sizeof(int);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("sum=%d\n",c);

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
