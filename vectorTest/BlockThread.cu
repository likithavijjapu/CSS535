#include "hip/hip_runtime.h"
# include <time.h>
# include <math.h>
# include <stdio.h>

__global__ void add( int *a , int *b , int *c)
{
	int index= threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] +b[index];
	
}

//# define N 125
#define thread_count 10

void random_ints(int* a, int h)
{

}

int main(void){
	int N;
	printf("\"Hello Vector !\"\n enter size of vector\n");
	scanf("%d",&N);
	int a[N],b[N],c[N];  // host copies of a, b,c
	int *d_a,*d_b,*d_c;  //// device copies of a, b, c

	int size = N * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	//setup input values for a, b, c  
	for ( int i=0;i<=N;i++)
  		{
    		a[i]=i+2;
    		b[i]=i+3;
   		c[i]=0;
  		}

	
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
	// start clocking
	clock_t start_time = clock(); 
	//Launch add() kernel on GPU
	add<<<N,thread_count>>>(d_a, d_b, d_c);
	hipDeviceSynchronize(); 
	//end clocking and measuring time for execution
	clock_t stop_time = clock();
	int time =stop_time - start_time;
	printf("time=%d\n", time);
	//Copy result back to host
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("c=");
	for(int i=0;i<N;i++){
	
	printf("%d+",c[i]);}
	printf("\n");

	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
